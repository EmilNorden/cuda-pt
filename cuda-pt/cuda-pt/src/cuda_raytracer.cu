#include "hip/hip_runtime.h"
#include "cuda_raytracer.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stack>

#include "Sphere.h"

#include "camera.h"

#include "intersectioninfo.h"
#include "opengl_surface.h"

#include <hiprand/hiprand_kernel.h>
#include <chrono>

__device__ void trace(Ray &ray, Sphere *spheres, int nSpheres, int max_depth, Vector3d &color, hiprandState &rand_state);
__device__ void shade(Ray &ray, const IntersectionInfo &ii, Sphere *spheres, int index, int nSpheres, int max_depth, Vector3d &color, hiprandState &rand_state);

__device__ bool intersect(Ray &ray, const Sphere &sphere) {
	double num1 = sphere.position.x() - ray.origin_.x();
	double num2 = sphere.position.y() - ray.origin_.y();
	double num3 = sphere.position.z() - ray.origin_.z();
	double num4 = (num1 * num1 + num2 * num2 + num3 * num3);
	double num5 = sphere.radius * sphere.radius;
	
	double num6 = (num1 * ray.direction_.x() + num2 * ray.direction_.y() + num3 * ray.direction_.z());

	if (num6 < 0.0)
	return false;
	double num7 = num4 - num6 * num6;
	if (num7 > num5)
		return false;

	return true;
}

__device__ bool intersect(Ray &ray, const Sphere &sphere, IntersectionInfo  &ii) {
	double num1 = sphere.position.x() - ray.origin_.x();
	double num2 = sphere.position.y() - ray.origin_.y();
	double num3 = sphere.position.z() - ray.origin_.z();
	double num4 = (num1 * num1 + num2 * num2 + num3 * num3);
	double num5 = sphere.radius * sphere.radius;
	
	double num6 = (num1 * ray.direction_.x() + num2 * ray.direction_.y() + num3 * ray.direction_.z());

	if (num6 < 0.0)
	return false;
	double num7 = num4 - num6 * num6;
	if (num7 > num5)
		return false;

	double num8 = sqrt(num5 - num7);

	ii.distance = abs(num6 - num8);
	ii.coordinate = ray.origin_ + (ray.direction_ * ii.distance);
	ii.surface_normal = ii.coordinate - sphere.position;

	ii.surface_normal.normalize();


	return true;
}

__device__ int get_intersected_sphere(Ray &ray, Sphere *spheres, int nSpheres)
{
	int intersection_index = -1;
	double best_dist = DBL_MAX;
	for(int i = 0; i < nSpheres; ++i)
	{
		IntersectionInfo temp_ii;
		if(intersect(ray, spheres[i], temp_ii) && temp_ii.distance < best_dist)
		{
			best_dist = temp_ii.distance;
			intersection_index = i;
		}
	}

	return intersection_index;
}


__device__ int get_intersected_sphere(Ray &ray, Sphere *spheres, int nSpheres, IntersectionInfo &ii)
{
	int intersection_index = -1;
	ii.distance = DBL_MAX;
	for(int i = 0; i < nSpheres; ++i)
	{
		IntersectionInfo temp_ii;
		if(intersect(ray, spheres[i], temp_ii) && temp_ii.distance < ii.distance)
		{
			ii = temp_ii;
			intersection_index = i;
		}
	}

	return intersection_index;
}

__device__ void shade_light(Sphere &light, const IntersectionInfo &ii, Sphere &object, Vector3d &color)
{
	Vector3d dir = light.position - ii.coordinate;
	double inv_square_length = 1.0 / dir.length_squared();
	dir.normalize();
	double dot = dir.dot(ii.surface_normal);
	if(dot < 0)
		dot = 0;
	color += light.emissive * object.diffuse * dot * inv_square_length * (1 - object.refl_coeff);
}

__device__ void pathtrace(Ray &ray, const IntersectionInfo &ii, int sphere_index, Sphere *spheres, int nSpheres, int max_depth, Vector3d &color, hiprandState &rand_state)
{
	Vector3d random_dir = Vector3d::rand_unit_in_hemisphere(ii.surface_normal, rand_state);
	Ray random_ray;
	random_ray.refractive_index = ray.refractive_index;
	random_ray.depth_ = ray.depth_ + 1;
	random_ray.origin_ = ii.coordinate;
	random_ray.direction_ = random_dir;

	IntersectionInfo random_ii;

	int random_sphere_index = get_intersected_sphere(random_ray, spheres, nSpheres, random_ii);
	if(random_sphere_index != -1)
	{
		Vector3d viewer = ray.direction_ * -1;
		Vector3d halfway = (random_dir + viewer);
		halfway.normalize();

		double factor = halfway.dot(ii.surface_normal);

		Vector3d random_color;
		shade(random_ray, random_ii, spheres, random_sphere_index, nSpheres, max_depth, random_color, rand_state);
		color += spheres[sphere_index].diffuse * random_color * pow(factor, 2);
	}
}

__device__ void shade(Ray &ray, const IntersectionInfo &ii, Sphere *spheres, int index, int nSpheres, int max_depth, Vector3d &color, hiprandState &rand_state)
{
	//printf("Depth %d\n", ray.depth_);
	if(ray.depth_ > max_depth)
		return;

	color += spheres[index].emissive;

	// Path tracing
	pathtrace(ray, ii, index, spheres, nSpheres, max_depth, color, rand_state);

	//// Diffuse
	for(int i = 0; i < nSpheres; ++i)
	{
		if(i != index && !spheres[i].emissive.is_zero())
		{
			Ray shadow_ray;
			shadow_ray.origin_ = ii.coordinate;
			shadow_ray.direction_ = spheres[i].position - ii.coordinate;
			shadow_ray.direction_.normalize();

			if (get_intersected_sphere(shadow_ray, spheres, nSpheres) == i) //(intersect(shadow_ray, spheres[i]))
			{
				shade_light(spheres[i], ii, spheres[index], color);
			}
		}
	}

	// Reflection

	if(spheres[index].refl_coeff > 0)
	{
		Ray reflected_ray;
		reflected_ray.refractive_index = ray.refractive_index;
		reflected_ray.depth_ = ray.depth_ + 1;
		reflected_ray.origin_ = ii.coordinate;
		reflected_ray.direction_ = ray.direction_ - ii.surface_normal * 2.0 * ray.direction_.dot(ii.surface_normal);
		reflected_ray.direction_.normalize();

		Vector3d refl_color;
		trace(reflected_ray, spheres, nSpheres, max_depth, refl_color, rand_state);
		color += refl_color * spheres[index].refl_coeff;
	}
}

__device__ void trace(Ray &ray, Sphere *spheres, int nSpheres, int max_depth, Vector3d &color, hiprandState &rand_state)
{
	IntersectionInfo ii;
	int sphere_index = get_intersected_sphere(ray, spheres, nSpheres, ii);

	if(sphere_index != -1)
	{
		shade(ray, ii, spheres, sphere_index, nSpheres, max_depth, color, rand_state);
	}
}

__global__ void raytrace_kernel(int *ptr, Camera *camera, int depth, int sample, Vector3d *buffer_d, hiprandState *rand_states, const Vector2i *resolution_d, Sphere **scene, int nSpheres)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	hiprandState &rand_state = rand_states[index_y * resolution_d->x() + index_x];
	Vector3d color;
	Ray ray;
	ray.depth_ = 0;
	camera->cast_perturbed_ray(ray, index_x, index_y, 2, rand_state);

	Sphere *spheres = *scene;

	trace(ray, spheres, nSpheres, depth, color, rand_state);

	color.clamp(Vector3d(0, 0, 0), Vector3d(1, 1, 1));

	Vector3d &current_color = buffer_d[index_y * (resolution_d->x()) + index_x];
	current_color.multiply(sample);
	current_color += color;
	current_color.multiply(1 / (double)(sample + 1));
	ptr[index_y * resolution_d->x() + index_x] = 255 << 24 | static_cast<int>(current_color.x() * 255) << 16  | static_cast<int>(current_color.y() * 255) << 8 | static_cast<int>(current_color.z() * 255);
}

hipError_t CudaRayTracer::render(Camera &camera, Sphere **scene, int nSpheres)
{
	dim3 block_size;
	block_size.x = 8;
	block_size.y = 8;

	dim3 grid_size;
	grid_size.x = surface_->resolution().x() / block_size.x;
	grid_size.y = surface_->resolution().y() / block_size.y;

	if(camera.updated_this_frame())
	{
		CUDA_CALL(hipMemcpy(camera_d, &camera, sizeof(Camera), hipMemcpyHostToDevice));
		current_sample_ = 0;
	}

	surface_->map();
	raytrace_kernel<<<grid_size, block_size>>>(static_cast<int*>(surface_->pixel_buffer_object_d()), camera_d, 4, current_sample_++, accumulation_buffer_d_, rand_state_d, surface_->resolution_d(), scene, nSpheres);
	hipError_t result = hipDeviceSynchronize();
	surface_->unmap();

	return result;
}

__device__ double camera_dist_d;
__global__ void get_camera_distance_kernel(Camera *camera, int x, int y, Sphere **scene, int nSpheres)
{
	Ray ray;
	camera->cast_ray(ray, x, y); 
	IntersectionInfo ii;

	camera_dist_d = DBL_MAX;
	if(get_intersected_sphere(ray, *scene, nSpheres, ii) != -1)
	{
		camera_dist_d = ii.distance;
	}
}

hipError_t CudaRayTracer::get_camera_distance(Camera &camera, const Vector2i &screen_coord, Sphere **scene, int n_spheres, double &dist_out)
{
	if(camera.updated_this_frame())
	{
		CUDA_CALL(hipMemcpy(camera_d, &camera, sizeof(Camera), hipMemcpyHostToDevice));
		current_sample_ = 0;
	}


	get_camera_distance_kernel<<<1, 1>>>(camera_d, screen_coord.x(), screen_coord.y(), scene, n_spheres);

	double camera_dist;
	void *camera_dist_device_ptr;
	
	hipMemcpyFromSymbol(&camera_dist, HIP_SYMBOL(camera_dist_device_ptr), sizeof(camera_dist), 0, hipMemcpyDeviceToHost);
	dist_out = camera_dist;
	return hipDeviceSynchronize();
}

void CudaRayTracer::set_surface(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(surface != nullptr && (surface_ == nullptr || surface->resolution() != surface_->resolution()))
	{
		init_curand(surface);
		init_accumulation_buffer(surface);
	}
	surface_ = surface;
}

__global__ void init_curand_kernel(hiprandState *state, unsigned long *seed, const Vector2i *resolution)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	hiprand_init(seed[index_y * resolution->x() + index_x], 0, 0, &state[index_y * resolution->x() + index_x]);
}

void CudaRayTracer::init_curand(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(rand_state_d != nullptr)
	{
		CUDA_CALL(hipFree(rand_state_d));
	}

	CUDA_CALL(hipMalloc(&rand_state_d, sizeof(hiprandState) * surface->resolution().x() * surface->resolution().y()));
	
	unsigned long *seeds_d;

	// Randomly generate seeds for the kernels hiprand states.
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::minstd_rand0 generator (seed);

	unsigned long *seeds_h = new unsigned long[surface->resolution().x() * surface->resolution().y()];
	for(int i = 0; i < surface->resolution().x() * surface->resolution().y(); ++i)
	{
		seeds_h[i] = generator();
	}

	CUDA_CALL(hipMalloc(&seeds_d, sizeof(unsigned long) * surface->resolution().x() * surface->resolution().y()));
	CUDA_CALL(hipMemcpy(seeds_d, seeds_h, sizeof(unsigned long) * surface->resolution().x() * surface->resolution().y(),  hipMemcpyHostToDevice));
	delete[] seeds_h;
	
	dim3 block_size;
	block_size.x = 4;
	block_size.y = 4;

	dim3 grid_size;
	grid_size.x = surface->resolution().x() / block_size.x;
	grid_size.y = surface->resolution().y() / block_size.y;

	init_curand_kernel<<<grid_size, block_size>>>(rand_state_d, seeds_d, surface->resolution_d());

	hipDeviceSynchronize();
}

CudaRayTracer::CudaRayTracer()
	: rand_state_d(nullptr), accumulation_buffer_d_(nullptr), current_sample_(0)
{
	CUDA_CALL(hipMalloc(&camera_d, sizeof(Camera)));
}

CudaRayTracer::~CudaRayTracer()
{
	CUDA_CALL(hipFree(camera_d));

	if(accumulation_buffer_d_ != nullptr)
	{
		CUDA_CALL(hipFree(accumulation_buffer_d_));
	}

	if(rand_state_d != nullptr)
	{
		CUDA_CALL(hipFree(rand_state_d));
	}
}

void CudaRayTracer::init_accumulation_buffer(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(accumulation_buffer_d_ != nullptr)
	{
		CUDA_CALL(hipFree(accumulation_buffer_d_));
	}

	CUDA_CALL(hipMalloc(&accumulation_buffer_d_, sizeof(Vector3d) * surface->resolution().x() * surface->resolution().y()));
}