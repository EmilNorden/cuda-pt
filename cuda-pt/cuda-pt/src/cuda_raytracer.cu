#include "hip/hip_runtime.h"
#include "cuda_raytracer.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include <stdio.h>
#include <stack>

#include "Sphere.h"

#include "camera.h"

#include "intersectioninfo.h"
#include "opengl_surface.h"

#include <hiprand/hiprand_kernel.h>
#include <chrono>

hipStream_t stream1, stream2, stream3, stream4;

__device__ void trace(const Ray &ray, const Sphere *spheres, const int nSpheres, const int max_depth, Vector3d &color, hiprandState &rand_state, const double branch_factor);
__device__ void shade(const Ray &ray, const IntersectionInfo &ii, const Sphere *spheres, const int index, const int nSpheres, const int max_depth, Vector3d &color, hiprandState &rand_state, const double branch_factor);
//
//__device__ bool intersect(Ray &ray, const Sphere &sphere, IntersectionInfo &ii) {
//	double a = ray.direction_.dot(ray.direction_);
//	double b = ray.direction_.dot((ray.origin_ - sphere.position) * 2.0);
//	double c = sphere.position.dot(sphere.position) + ray.origin_.dot(ray.origin_) - (ray.origin_.dot(sphere.position) * 2.0) - sphere.radius * sphere.radius;
//	double D = b * b + (-4.0) * a * c;
//
//	if(D < 0)
//		return false;
//
//	D = sqrt(D);
//
//	double t = (-0.5)*(b+D)/a;
//	if(t > 0.0)
//	{
//		ii.distance = sqrt(a)*t;
//		ii.coordinate = ray.origin_ + ray.direction_*t;
//		ii.surface_normal = (ii.coordinate - sphere.position) / sphere.radius;
//	}
//	else
//	{
//		return false;
//	}
//
//	return true;
//}

__device__ bool intersect(const Ray &ray, const Sphere &sphere, IntersectionInfo  &ii) {
	double num1 = __dsub_rn(sphere.position.x(), ray.origin_.x());
	double num2 = __dsub_rn(sphere.position.y(), ray.origin_.y());
	double num3 = __dsub_rn(sphere.position.z(), ray.origin_.z());
	double num4 = __fma_rn(num1, num1, __fma_rn(num2, num2, __dmul_rn(num3, num3)));
	double num5 = __dmul_rn(sphere.radius, sphere.radius);
	
	double num6 = __fma_rn(num1, ray.direction_.x(), 
					__fma_rn(num2, ray.direction_.y(), __dmul_rn(num3, ray.direction_.z())));

	if (num6 < 0.0)
	return false;
	double num7 = num4 - num6 * num6;
	if (num7 > num5)
		return false;

	ii.distance = abs(num6 - sqrt(num5 - num7));
	ii.coordinate = ray.origin_ + (ray.direction_ * ii.distance);
	ii.surface_normal = ii.coordinate - sphere.position;

	ii.surface_normal.normalize_device();


	return true;
}

__device__ int get_intersected_sphere(const Ray &ray, const Sphere *spheres, const int nSpheres)
{
	int intersection_index = -1;
	double best_dist = DBL_MAX;
	for(int i = 0; i < nSpheres; ++i)
	{
		IntersectionInfo temp_ii;
		if(intersect(ray, spheres[i], temp_ii) && temp_ii.distance < best_dist)
		{
			best_dist = temp_ii.distance;
			intersection_index = i;
		}
	}

	return intersection_index;
}


__device__ int get_intersected_sphere(const Ray &ray, const Sphere *spheres, const int nSpheres, IntersectionInfo &ii)
{
	int intersection_index = -1;
	ii.distance = DBL_MAX;
	for(int i = 0; i < nSpheres; ++i)
	{
		IntersectionInfo temp_ii;
		if(intersect(ray, spheres[i], temp_ii) && temp_ii.distance < ii.distance)
		{
			ii = temp_ii;
			intersection_index = i;
		}
	}

	return intersection_index;
}

__device__ void shade_light(const Sphere &light, const IntersectionInfo &ii, const Sphere &object, Vector3d &color)
{
	Vector3d dir = light.position - ii.coordinate;
	double inv_square_length = 1.0 / dir.length_squared();
	dir.normalize_device();
	double dot = dir.dot_device(ii.surface_normal);
	if(dot < 0)
		dot = 0;
	color += light.emissive * object.diffuse * dot * inv_square_length;
}

__device__ void pathtrace(const Ray &ray, const IntersectionInfo &ii, const int sphere_index, const Sphere *spheres, const int nSpheres, const int max_depth, Vector3d &color, hiprandState &rand_state, const double branch_factor)
{
	Vector3d random_dir = Vector3d::rand_unit_in_hemisphere(ii.surface_normal, rand_state);
	Ray random_ray;
	random_ray.refractive_index = ray.refractive_index;
	random_ray.depth_ = ray.depth_ + 1;
	random_ray.origin_ = ii.coordinate;
	random_ray.direction_ = random_dir;

	IntersectionInfo random_ii;

	int random_sphere_index = get_intersected_sphere(random_ray, spheres, nSpheres, random_ii);
	if(random_sphere_index != -1)
	{
		Vector3d viewer = ray.direction_ * -1;
		Vector3d halfway = (random_dir + viewer);
		halfway.normalize_device();

		double factor = halfway.dot_device(ii.surface_normal);

		Vector3d random_color;
		shade(random_ray, random_ii, spheres, random_sphere_index, nSpheres, max_depth, random_color, rand_state, branch_factor);
		color += spheres[sphere_index].diffuse * random_color * pow(factor, 2);
	}
}

__device__ void shade(const Ray &ray, const IntersectionInfo &ii, const Sphere *spheres, const int index, const int nSpheres, const int max_depth, Vector3d &color, hiprandState &rand_state, const double branch_factor)
{
	if(ray.depth_ > max_depth)
		return;

	color += spheres[index].emissive;

	if(branch_factor > spheres[index].refl_coeff)
	{
		// Path tracing
		pathtrace(ray, ii, index, spheres, nSpheres, max_depth, color, rand_state, branch_factor);
		//// Diffuse
		for(int i = 0; i < nSpheres; ++i)
		{
			if(i != index && !spheres[i].emissive.is_zero())
			{
				Ray shadow_ray;
				shadow_ray.origin_ = ii.coordinate;
				shadow_ray.direction_ = spheres[i].position - ii.coordinate;
				shadow_ray.direction_.normalize_device();

				if (get_intersected_sphere(shadow_ray, spheres, nSpheres) == i)
				{
					shade_light(spheres[i], ii, spheres[index], color);
				}
			}
		}
	}
	else
	{
		// Reflection
		Ray reflected_ray;
		reflected_ray.refractive_index = ray.refractive_index;
		reflected_ray.depth_ = ray.depth_ + 1;
		reflected_ray.origin_ = ii.coordinate;
		reflected_ray.direction_ = ray.direction_ - ii.surface_normal * 2.0 * ray.direction_.dot_device(ii.surface_normal);
		reflected_ray.direction_.normalize_device();

		Vector3d refl_color;
		trace(reflected_ray, spheres, nSpheres, max_depth, refl_color, rand_state, branch_factor);
		color += refl_color;
	}
}

__device__ void trace(const Ray &ray, const Sphere *spheres, const int nSpheres, const int max_depth, Vector3d &color, hiprandState &rand_state, const double branch_factor)
{
	IntersectionInfo ii;
	int sphere_index = get_intersected_sphere(ray, spheres, nSpheres, ii);

	if(sphere_index != -1)
	{
		shade(ray, ii, spheres, sphere_index, nSpheres, max_depth, color, rand_state, branch_factor);
	}
}

__global__ void raytrace_kernel(int *ptr, Camera *camera, int depth, int sample, Vector3d *buffer_d, hiprandState *rand_states, const Vector2i *resolution_d, Sphere **scene, int nSpheres, int y_offset)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	index_y += y_offset;

	hiprandState &rand_state = rand_states[index_y * resolution_d->x() + index_x];
	Vector3d color;
	Ray ray;
	ray.depth_ = 0;
	camera->cast_perturbed_ray(ray, index_x, index_y, 0.125, rand_state);

	Sphere *spheres = *scene;

	double branch_factor = hiprand_uniform(&rand_state);

	trace(ray, spheres, nSpheres, depth, color, rand_state, branch_factor);

	color.clamp(Vector3d(0, 0, 0), Vector3d(1, 1, 1));

	Vector3d &current_color = buffer_d[index_y * (resolution_d->x()) + index_x];
	current_color.multiply(sample);
	current_color += color;
	current_color.multiply(__drcp_rn((double)(sample + 1)));
	ptr[index_y * resolution_d->x() + index_x] = 255 << 24 | static_cast<int>(current_color.x() * 255) << 16  | static_cast<int>(current_color.y() * 255) << 8 | static_cast<int>(current_color.z() * 255);
}

hipError_t CudaRayTracer::render(Camera &camera, Sphere **scene, int nSpheres)
{
	dim3 block_size;
	block_size.x = 8;
	block_size.y = 8;

	dim3 grid_size;
	grid_size.x = surface_->resolution().x() / block_size.x;
	grid_size.y = (surface_->resolution().y() / 1) / block_size.y;

	if(camera.updated_this_frame())
	{
		CUDA_CALL(hipMemcpy(camera_d, &camera, sizeof(Camera), hipMemcpyHostToDevice));
		current_sample_ = 0;
	}

	surface_->map();
	raytrace_kernel<<<grid_size, block_size, 0, stream1>>>(static_cast<int*>(surface_->pixel_buffer_object_d()), camera_d, 4, current_sample_++, accumulation_buffer_d_, rand_state_d, surface_->resolution_d(), scene, nSpheres, 0);
	//raytrace_kernel<<<grid_size, block_size, 0, stream2>>>(static_cast<int*>(surface_->pixel_buffer_object_d()), camera_d, 4, current_sample_++, accumulation_buffer_d_, rand_state_d, surface_->resolution_d(), scene, nSpheres, 240);
	//raytrace_kernel<<<grid_size, block_size, 0, stream3>>>(static_cast<int*>(surface_->pixel_buffer_object_d()), camera_d, 4, current_sample_, accumulation_buffer_d_, rand_state_d, surface_->resolution_d(), scene, nSpheres, 240);
	//raytrace_kernel<<<grid_size, block_size, 0, stream4>>>(static_cast<int*>(surface_->pixel_buffer_object_d()), camera_d, 4, current_sample_++, accumulation_buffer_d_, rand_state_d, surface_->resolution_d(), scene, nSpheres, 360);
	hipError_t result = hipDeviceSynchronize();
	surface_->unmap();

	return result;
}
double *distance_d = nullptr;
__device__ double camera_dist_d;
__global__ void get_camera_distance_kernel(Camera *camera, int x, int y, Sphere **scene, int nSpheres, double *distance_d)
{
	Ray ray;
	camera->cast_ray(ray, x, y); 
	IntersectionInfo ii;

	camera_dist_d = DBL_MAX;
	if(get_intersected_sphere(ray, *scene, nSpheres, ii) != -1)
	{
		*distance_d = ii.distance;
	}
}


hipError_t CudaRayTracer::get_camera_distance(Camera &camera, const Vector2i &screen_coord, Sphere **scene, int n_spheres, double &dist_out)
{
	if(camera.updated_this_frame())
	{
		CUDA_CALL(hipMemcpy(camera_d, &camera, sizeof(Camera), hipMemcpyHostToDevice));
		current_sample_ = 0;
	}

	if(!distance_d)
	{
		hipMalloc(&distance_d, sizeof(double));
	}

	get_camera_distance_kernel<<<1, 1>>>(camera_d, screen_coord.x(), screen_coord.y(), scene, n_spheres, distance_d);

	hipMemcpy(&dist_out, distance_d, sizeof(double), hipMemcpyDeviceToHost);

	return hipDeviceSynchronize();
}

void CudaRayTracer::set_surface(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(surface != nullptr && (surface_ == nullptr || surface->resolution() != surface_->resolution()))
	{
		init_curand(surface);
		init_accumulation_buffer(surface);
	}
	surface_ = surface;
}

__global__ void init_curand_kernel(hiprandState *state, unsigned long *seed, const Vector2i *resolution)
{
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;

	hiprand_init(seed[index_y * resolution->x() + index_x], 0, 0, &state[index_y * resolution->x() + index_x]);
}

void CudaRayTracer::init_curand(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(rand_state_d != nullptr)
	{
		CUDA_CALL(hipFree(rand_state_d));
	}

	CUDA_CALL(hipMalloc(&rand_state_d, sizeof(hiprandState) * surface->resolution().x() * surface->resolution().y()));
	
	unsigned long *seeds_d;

	// Randomly generate seeds for the kernels hiprand states.
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::minstd_rand0 generator (seed);

	unsigned long *seeds_h = new unsigned long[surface->resolution().x() * surface->resolution().y()];
	for(int i = 0; i < surface->resolution().x() * surface->resolution().y(); ++i)
	{
		seeds_h[i] = generator();
	}

	CUDA_CALL(hipMalloc(&seeds_d, sizeof(unsigned long) * surface->resolution().x() * surface->resolution().y()));
	CUDA_CALL(hipMemcpy(seeds_d, seeds_h, sizeof(unsigned long) * surface->resolution().x() * surface->resolution().y(),  hipMemcpyHostToDevice));
	delete[] seeds_h;
	
	dim3 block_size;
	block_size.x = 4;
	block_size.y = 4;

	dim3 grid_size;
	grid_size.x = surface->resolution().x() / block_size.x;
	grid_size.y = surface->resolution().y() / block_size.y;

	init_curand_kernel<<<grid_size, block_size>>>(rand_state_d, seeds_d, surface->resolution_d());

	hipDeviceSynchronize();
}

CudaRayTracer::CudaRayTracer()
	: rand_state_d(nullptr), accumulation_buffer_d_(nullptr), current_sample_(0)
{
	CUDA_CALL(hipMalloc(&camera_d, sizeof(Camera)));

	CUDA_CALL(hipStreamCreate(&stream1));
	CUDA_CALL(hipStreamCreate(&stream2));
	CUDA_CALL(hipStreamCreate(&stream3));
	CUDA_CALL(hipStreamCreate(&stream4));
}

CudaRayTracer::~CudaRayTracer()
{
	CUDA_CALL(hipFree(camera_d));

	if(accumulation_buffer_d_ != nullptr)
	{
		CUDA_CALL(hipFree(accumulation_buffer_d_));
	}

	if(rand_state_d != nullptr)
	{
		CUDA_CALL(hipFree(rand_state_d));
	}
}

void CudaRayTracer::init_accumulation_buffer(const std::shared_ptr<OpenGLSurface> &surface)
{
	if(accumulation_buffer_d_ != nullptr)
	{
		CUDA_CALL(hipFree(accumulation_buffer_d_));
	}

	CUDA_CALL(hipMalloc(&accumulation_buffer_d_, sizeof(Vector3d) * surface->resolution().x() * surface->resolution().y()));
	CUDA_CALL(hipMemset(accumulation_buffer_d_, 0, sizeof(Vector3d) * surface->resolution().x() * surface->resolution().y()));
}